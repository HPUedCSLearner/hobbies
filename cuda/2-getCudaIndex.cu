
#include <hip/hip_runtime.h>
#include <stdio.h>


__global__ void printHelloCuda_1()
{
    // printf("gridDim[%d, %d, %d]\n", gridDim.x, gridDim.y, gridDim.z);
    printf("blockIdx[%d, %d, %d]\n", blockIdx.x, blockIdx.y, blockIdx.z);

    // printf("blockDim[%d, %d, %d]\n", blockDim.x, blockDim.y, blockDim.z);
    printf("threadIdx[%d, %d, %d]\n", threadIdx.x, threadIdx.y, threadIdx.z);


    int globalIndexX = blockIdx.x * blockDim.x + threadIdx.x;
    int globalIndexY = blockIdx.y * blockDim.y + threadIdx.y;
    int globalIndexZ = blockIdx.z * blockDim.z + threadIdx.z;

    printf("globalIndex[%d, %d, %d]\n", globalIndexX, globalIndexY, globalIndexZ);

    int globalIndex = globalIndexZ * gridDim.y * gridDim.x * blockDim.z * blockDim.y * blockDim.x +
                      globalIndexY * gridDim.x * blockDim.y * blockDim.x +
                      globalIndexX * blockDim.x;
    
    printf("globalIndex[%d]\n", globalIndex);
}

__global__ void printHelloCuda_2()
{
    int globalIndexX = blockIdx.x * blockDim.x + threadIdx.x;
    int globalIndexY = blockIdx.y * blockDim.y + threadIdx.y;
    int globalIndexZ = blockIdx.z * blockDim.z + threadIdx.z;


    // 这个 globalIndex 为什么要除以2 才是想要全局索引？？？
    int globalIndex = globalIndexZ * gridDim.y * gridDim.x * blockDim.z * blockDim.y * blockDim.x +
                      globalIndexY * gridDim.x * blockDim.y * blockDim.x +
                      globalIndexX * blockDim.x;
    

    printf("<<<grids, block>>> Info : {gridDim[%d, %d, %d], blockDim[%d, %d, %d]}\n", gridDim.x, gridDim.y, gridDim.z, blockDim.x, blockDim.y, blockDim.z);
    
    printf("I am globalIndex[%d], globalIndex[%d, %d, %d], in blockIdx[%d, %d, %d], in threadIdx[%d, %d, %d]\n", 
            globalIndex / 2, globalIndexX, globalIndexY, globalIndexZ, 
            blockIdx.x, blockIdx.y, blockIdx.z,
            threadIdx.x, threadIdx.y, threadIdx.z);
}


__global__ void printHelloCuda_3()
{
    int globalIndexX = blockIdx.x * blockDim.x + threadIdx.x;
    int globalIndexY = blockIdx.y * blockDim.y + threadIdx.y;
    int globalIndexZ = blockIdx.z * blockDim.z + threadIdx.z;


    // 求全局索引公式: 全局索引 = blockNum * blockSize + threadNum(in block)

    // 将block索引坐标 （blockIdx.x, blockIdx.y, blockIdx.z) 转成 一维block索引
    int blockNum  = blockIdx.z * (gridDim.x * gridDim.y) + blockIdx.y * (gridDim.x) + blockIdx.x;

    // 通过block的维度，求一个block有多少个线程
    int blockSize = blockDim.x * blockDim.y * blockDim.z;

    // 将thread索引坐标 [threadIdx.x, threadIdx.y, threadIdx.z] 转成 一维thread索引
    int threadNum = threadIdx.z * (blockDim.x * blockDim.y) + threadIdx.y * (blockDim.x) + threadIdx.x;

    // 求全局索引公式: 全局索引 = blockNum * blockSize + threadNum(in block) // {我这个线程是在第几个block * blocksize + 我这个线程在这个block的索引}
    int globalIndex = blockNum * blockSize + threadNum;
    

    printf("<<<grids, block>>> Info : {gridDim[%d, %d, %d], blockDim[%d, %d, %d]}\n", gridDim.x, gridDim.y, gridDim.z, blockDim.x, blockDim.y, blockDim.z);
    
    printf("I am globalIndex[%d], globalIndex[%d, %d, %d], in blockIdx[%d, %d, %d], in threadIdx[%d, %d, %d]\n", 
            globalIndex, globalIndexX, globalIndexY, globalIndexZ, 
            blockIdx.x, blockIdx.y, blockIdx.z,
            threadIdx.x, threadIdx.y, threadIdx.z);
}

__global__ void printHelloCuda_4()
{
    int globalIndexX = blockIdx.x * blockDim.x + threadIdx.x;
    int globalIndexY = blockIdx.y * blockDim.y + threadIdx.y;
    int globalIndexZ = blockIdx.z * blockDim.z + threadIdx.z;

    int globalDimX = gridDim.x * blockDim.x;
    int globalDimY = gridDim.y * blockDim.y;
    int globalDimZ = gridDim.z * blockDim.z;


    // 求全局索引原理：把grid和blocks看作是一个大的cube
    // thread的坐标是[globalIndexX, globalIndexY, globalIndexZ]
    // 这个大的cube的维度是[globalDimX, globalDimY, globalDimZ]

    int globalIndex = globalIndexZ * (globalDimY * globalDimX) + globalIndexY * (globalDimX) + globalIndexX;
    

    printf("in printHelloCuda_4, <<<grids, block>>> Info : {gridDim[%d, %d, %d], blockDim[%d, %d, %d]}\n", gridDim.x, gridDim.y, gridDim.z, blockDim.x, blockDim.y, blockDim.z);
    
    printf("I am globalIndex[%d], globalIndex[%d, %d, %d], in blockIdx[%d, %d, %d], in threadIdx[%d, %d, %d]\n", 
            globalIndex, globalIndexX, globalIndexY, globalIndexZ, 
            blockIdx.x, blockIdx.y, blockIdx.z,
            threadIdx.x, threadIdx.y, threadIdx.z);
}


int main() {


    // printHelloCuda_3<<<3, 1>>>();
    // printHelloCuda_3<<<1, 3>>>();

    dim3 grids(2,2,1);
    dim3 blocks(2,1,1);
    // printHelloCuda_1<<<grids, blocks>>>();
    // printHelloCuda_2<<<grids, blocks>>>();
    // printHelloCuda_3<<<grids, blocks>>>();
    printHelloCuda_4<<<grids, blocks>>>();


     // 检查核函数执行是否失败
    hipError_t cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "Kernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        return 1;
    }

    hipDeviceSynchronize();

    return 0;
}

// blockIdx     与 gridDim  相对应
// threadIdx    与 blockDim 相对应

// <<<grids, block>>> Info : {gridDim[2, 1, 1], blockDim[2, 1, 1]}
// <<<grids, block>>> Info : {gridDim[2, 1, 1], blockDim[2, 1, 1]}
// <<<grids, block>>> Info : {gridDim[2, 1, 1], blockDim[2, 1, 1]}
// <<<grids, block>>> Info : {gridDim[2, 1, 1], blockDim[2, 1, 1]}
// I am globalIndex[0], globalIndex[0, 0, 0], in blockIdx[0, 0, 0], in threadIdx[0, 0, 0]
// I am globalIndex[1], globalIndex[1, 0, 0], in blockIdx[0, 0, 0], in threadIdx[1, 0, 0]
// I am globalIndex[2], globalIndex[2, 0, 0], in blockIdx[1, 0, 0], in threadIdx[0, 0, 0]
// I am globalIndex[3], globalIndex[3, 0, 0], in blockIdx[1, 0, 0], in threadIdx[1, 0, 0]




// <<<grids, block>>> Info : {gridDim[2, 2, 1], blockDim[2, 1, 1]}
// <<<grids, block>>> Info : {gridDim[2, 2, 1], blockDim[2, 1, 1]}
// <<<grids, block>>> Info : {gridDim[2, 2, 1], blockDim[2, 1, 1]}
// <<<grids, block>>> Info : {gridDim[2, 2, 1], blockDim[2, 1, 1]}
// <<<grids, block>>> Info : {gridDim[2, 2, 1], blockDim[2, 1, 1]}
// <<<grids, block>>> Info : {gridDim[2, 2, 1], blockDim[2, 1, 1]}
// <<<grids, block>>> Info : {gridDim[2, 2, 1], blockDim[2, 1, 1]}
// <<<grids, block>>> Info : {gridDim[2, 2, 1], blockDim[2, 1, 1]}
// I am globalIndex[4], globalIndex[2, 1, 0], in blockIdx[1, 1, 0], in threadIdx[0, 0, 0]
// I am globalIndex[5], globalIndex[3, 1, 0], in blockIdx[1, 1, 0], in threadIdx[1, 0, 0]
// I am globalIndex[0], globalIndex[0, 0, 0], in blockIdx[0, 0, 0], in threadIdx[0, 0, 0]
// I am globalIndex[1], globalIndex[1, 0, 0], in blockIdx[0, 0, 0], in threadIdx[1, 0, 0]
// I am globalIndex[2], globalIndex[0, 1, 0], in blockIdx[0, 1, 0], in threadIdx[0, 0, 0]
// I am globalIndex[3], globalIndex[1, 1, 0], in blockIdx[0, 1, 0], in threadIdx[1, 0, 0]
// I am globalIndex[2], globalIndex[2, 0, 0], in blockIdx[1, 0, 0], in threadIdx[0, 0, 0]
// I am globalIndex[3], globalIndex[3, 0, 0], in blockIdx[1, 0, 0], in threadIdx[1, 0, 0]