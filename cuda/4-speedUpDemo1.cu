
#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdlib>
#include <ctime>

// CPU 串行版本的向量加法
void addVectorCPU(int* a, int* b, int* c, int size) {
    for (int i = 0; i < size; ++i) {
        c[i] = a[i] + b[i];
    }
}

// GPU 版本的向量加法
__global__ void addVectorGPU(int* a, int* b, int* c, int size) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < size) {
        c[tid] = a[tid] + b[tid];
    }
}

int main() {
    const int size = 1000000;
    const int threadsPerBlock = 256;
    const int blocksPerGrid = (size + threadsPerBlock - 1) / threadsPerBlock;

    // 分配和初始化向量
    int* a = new int[size];
    int* b = new int[size];
    int* c_cpu = new int[size];
    int* c_gpu = new int[size];

    srand(static_cast<unsigned>(time(0)));
    for (int i = 0; i < size; ++i) {
        a[i] = rand() % 100;
        b[i] = rand() % 100;
    }

    // CPU 串行版本的向量加法
    clock_t start_cpu = clock();
    addVectorCPU(a, b, c_cpu, size);
    clock_t end_cpu = clock();
    double cpu_time = static_cast<double>(end_cpu - start_cpu) / CLOCKS_PER_SEC;

    // GPU 版本的向量加法
    int* d_a, *d_b, *d_c;
    hipMallocManaged((void**)&d_a, size * sizeof(int));
    hipMallocManaged((void**)&d_b, size * sizeof(int));
    hipMallocManaged((void**)&d_c, size * sizeof(int));
    //cudaMalloc((void**)&d_a, size * sizeof(int));
    //cudaMalloc((void**)&d_b, size * sizeof(int));
    //cudaMalloc((void**)&d_c, size * sizeof(int));

    hipMemcpy(d_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size * sizeof(int), hipMemcpyHostToDevice);

    clock_t start_gpu = clock();
    addVectorGPU<<<blocksPerGrid, threadsPerBlock>>>(d_a, d_b, d_c, size);
    hipDeviceSynchronize();
    clock_t end_gpu = clock();
    double gpu_time = static_cast<double>(end_gpu - start_gpu) / CLOCKS_PER_SEC;

    hipMemcpy(c_gpu, d_c, size * sizeof(int), hipMemcpyDeviceToHost);

    // 检查结果是否一致
    for (int i = 0; i < size; ++i) {
        if (c_cpu[i] != c_gpu[i]) {
            std::cerr << "Error: CPU and GPU results do not match!" << std::endl;
            break;
        }
    }

    // 计算加速比
    double speedup = cpu_time / gpu_time;

    std::cout << "CPU Time: " << cpu_time << " seconds" << std::endl;
    std::cout << "GPU Time: " << gpu_time << " seconds" << std::endl;
    std::cout << "Speedup: " << speedup << "x" << std::endl;

    // 释放内存
    delete[] a;
    delete[] b;
    delete[] c_cpu;
    delete[] c_gpu;

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return 0;
}
