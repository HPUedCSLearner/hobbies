
#include <hip/hip_runtime.h>
#include <stdio.h>

int main() {
    int deviceCount;
    hipGetDeviceCount(&deviceCount);

    for (int i = 0; i < deviceCount; ++i) {
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, i);

        printf("Device %d:\n", i);
        printf("  Name: %s\n", deviceProp.name);
        printf("  Max threads per block: %d\n", deviceProp.maxThreadsPerBlock);
        printf("  Max threads per multiprocessor: %d\n", deviceProp.maxThreadsPerMultiProcessor);
        // 可以查询其他属性

        printf("\n");
    }

    return 0;
}


// ==========A 100============
// Device 0:
//   Name: NVIDIA A100 80GB PCIe
//   Max threads per block: 1024
//   Max threads per multiprocessor: 2048