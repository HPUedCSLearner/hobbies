
#include <hip/hip_runtime.h>
#include<string.h>
#include<sys/time.h>
#include<stdio.h>


#define N (8)

__global__ void add_kernal(int *a, int *b, int *c, int size) 
{

    int globalIndexX = blockIdx.x * blockDim.x + threadIdx.x;
    int globalIndexY = blockIdx.y * blockDim.y + threadIdx.y;
    int globalIndexZ = blockIdx.z * blockDim.z + threadIdx.z;

    int globalIndex = globalIndexZ * size * size + globalIndexY * size + globalIndexX;
    c[globalIndex] = a[globalIndex] + b[globalIndex] + 1;

}

int main()
{

    struct timeval start, end;
    double c_time_use;
    double g_time_use;

    int a[N][N][N][N][N][N];
    int b[N][N][N][N][N][N];
    int c[N][N][N][N][N][N];
    int C[N][N][N][N][N][N];

    memset(a, 0, sizeof(a));
    memset(b, 0, sizeof(b));
    memset(c, 0, sizeof(c));

    printf("a has %ld bytes\n", sizeof(a) * 4);
    printf("b has %ld bytes\n", sizeof(b) * 4);
    printf("c has %ld bytes\n", sizeof(c) * 4);



    // cudaMalloc()
    int *d_a, *d_b, *d_c;
    // cudaMallocManaged((void**)&d_a, N*N*N*N*N*N*sizeof(int));
    hipMalloc((void**)&d_a, N*N*N*N*N*N*sizeof(int));
    hipMalloc((void**)&d_b, N*N*N*N*N*N*sizeof(int));
    hipMalloc((void**)&d_c, N*N*N*N*N*N*sizeof(int));

    hipMemcpy(d_a, a, N*N*N*N*N*N*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, N*N*N*N*N*N*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_c, c, N*N*N*N*N*N*sizeof(int), hipMemcpyHostToDevice);

    // printf("测试主机可否访问，设备内存%d\n", d_a[0][0][0][0][0][0]); // expression must have pointer-to-object type but it has type "int"

    // CPU 运算
    gettimeofday(&start, NULL);

                        for(int n = 0; n < N; ++n) {
                    for(int m = 0; m < N; ++m) {
                for(int l = 0; l < N; ++l) {
            for(int k = 0; k < N; ++k) {
        for(int j = 0; j < N; ++j) {
    for(int i = 0; i < N; ++i) {
                            c[i][j][k][l][m][n] = a[i][j][k][l][m][n] + b[i][j][k][l][m][n] + 1;
                        }
                    }
                }

            }
        }
    }
    gettimeofday(&end, NULL);


    c_time_use = (end.tv_sec-start.tv_sec) + (end.tv_usec-start.tv_usec) / 1e6;

    printf("cpu time ues %lf\n", c_time_use);

    // check
    for(int i = 0; i < N; ++i) {
        for(int j = 0; j < N; ++j) {
            for(int k = 0; k < N; ++k) {
                for(int l = 0; l < N; ++l) {
                    for(int m = 0; m < N; ++m) {
                        for(int n = 0; n < N; ++n) {
                            if ( c[i][j][k][l][m][n]  != 1) {
                                printf("cal error, expect 1, error at (%d,%d,%d,%d,%d,%d)", i, j, k, l, m, n);
                                break;
                            }
                        }
                    }

                }

            }
        }
    }

    // GPU 运算
    dim3 blocks(N, N, N);
    dim3 grids(N,N,N);

    gettimeofday(&start, NULL);
    add_kernal<<<grids, blocks>>>(d_a, d_b, d_c, N);
    hipDeviceSynchronize();
    gettimeofday(&end, NULL);

    g_time_use = (end.tv_sec-start.tv_sec) + (end.tv_usec-start.tv_usec) / 1e6;

    printf("gpu time ues %lf\n", g_time_use);

    hipMemcpy(C, d_c, N*N*N*N*N*N*sizeof(int), hipMemcpyDeviceToHost);

    // check
    for(int i = 0; i < N; ++i) {
        for(int j = 0; j < N; ++j) {
            for(int k = 0; k < N; ++k) {
                for(int l = 0; l < N; ++l) {
                    for(int m = 0; m < N; ++m) {
                        for(int n = 0; n < N; ++n) {
                            if ( C[i][j][k][l][m][n]  != 1) {
                                printf("cal error, expect 1, error at (%d,%d,%d,%d,%d,%d)\n", i, j, k, l, m, n);
                                break;
                            }
                            break;
                        }
                        break;
                    }
                    break;
                }
                break;
            }
            break;
        }
        break;
    }

    printf("speed up: %lf\n", c_time_use / g_time_use);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return 0;
}